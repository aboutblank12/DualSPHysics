#include "hip/hip_runtime.h"
//HEAD_DSPH
/*
 <DUALSPHYSICS>  Copyright (c) 2016, Dr Jose M. Dominguez et al. (see http://dual.sphysics.org/index.php/developers/). 

 EPHYSLAB Environmental Physics Laboratory, Universidade de Vigo, Ourense, Spain.
 School of Mechanical, Aerospace and Civil Engineering, University of Manchester, Manchester, U.K.

 This file is part of DualSPHysics. 

 DualSPHysics is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by
 the Free Software Foundation, either version 3 of the License, or (at your option) any later version. 

 DualSPHysics is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details. 

 You should have received a copy of the GNU General Public License, along with DualSPHysics. If not, see <http://www.gnu.org/licenses/>. 
*/

/// \file JSphGpuSimple_ker.cu \brief Implements functions and CUDA kernels for the Particle Interaction and System Update.

#include "JSphGpuSimple_ker.h"
//#include "Functions.h"
//#include "FunctionsCuda.h"
//#include <hip/hip_math_constants.h>
//#include "JDgKerPrint.h"
//#include "JDgKerPrint_ker.h"


namespace cusphs{
#include "FunctionsBasic_iker.cu"


//##############################################################################
//# Kernels to prepare data before Interaction_Forces().
//##############################################################################
//------------------------------------------------------------------------------
/// Update PosCellg[] according to current position of particles.
/// Actualiza PosCellg[] segun la posicion de las particulas.
//------------------------------------------------------------------------------
__global__ void KerUpdatePosCell(unsigned np,double3 posmin,float dosh
  ,const double2 *posxy,const double *posz,float4 *poscell)
{
  const unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<np){
    const double2 rxy=posxy[p];
    const double dx=rxy.x-posmin.x;
    const double dy=rxy.y-posmin.y;
    const double dz=posz[p]-posmin.z;
    const unsigned cx=unsigned(dx/dosh);
    const unsigned cy=unsigned(dy/dosh);
    const unsigned cz=unsigned(dz/dosh);
    const float px=float(dx-(double(dosh)*cx));
    const float py=float(dy-(double(dosh)*cy));
    const float pz=float(dz-(double(dosh)*cz));
    const float pw=__uint_as_float(CEL_Code(cx,cy,cz));
    poscell[p]=make_float4(px,py,pz,pw);
  }
}
//==============================================================================
/// Update PosCellg[] according to current position of particles.
/// Actualiza PosCellg[] segun la posicion de las particulas.
//==============================================================================
void UpdatePosCell(unsigned np,tdouble3 posmin,float dosh
  ,const double2 *posxy,const double *posz,float4 *poscell,hipStream_t stm)
{
  const dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
  if(np)KerUpdatePosCell <<<sgrid,SPHBSIZE,0,stm>>> (np,Double3(posmin),dosh,posxy,posz,poscell);
}

//------------------------------------------------------------------------------
/// Initialises ace array with 0 for bound and gravity for fluid.
/// Inicializa el array ace con 0 para contorno y gravity para fluido.
//------------------------------------------------------------------------------
__global__ void KerInitAceGravity(unsigned np,unsigned npb,float3 gravity,float3 *ace)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<np){
    ace[p]=(p<npb? make_float3(0,0,0): gravity);
  }
}
//==============================================================================
/// Initialises ace array with 0 for bound and gravity for fluid.
/// Inicializa el array ace con 0 para contorno y gravity para fluido.
//==============================================================================
void InitAceGravity(unsigned np,unsigned npb,tfloat3 gravity,float3 *ace,hipStream_t stm){
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    KerInitAceGravity <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,Float3(gravity),ace);
  }
}


//##############################################################################
//# Kernels to run after Interaction_Forces().
//##############################################################################
//------------------------------------------------------------------------------
/// Sets v[].y to zero.
/// Pone v[].y a cero.
//------------------------------------------------------------------------------
__global__ void KerResety(unsigned n,unsigned ini,float3 *v)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n)v[p+ini].y=0;
}
//==============================================================================
/// Sets v[].y to zero.
/// Pone v[].y a cero.
//==============================================================================
void Resety(unsigned n,unsigned ini,float3 *v,hipStream_t stm){
  if(n){
    dim3 sgrid=GetSimpleGridSize(n,SPHBSIZE);
    KerResety <<<sgrid,SPHBSIZE,0,stm>>> (n,ini,v);
  }
}


//##############################################################################
//# Kernels for ComputeStep (vel & rhop).
//# Kernels para ComputeStep (vel & rhop).
//##############################################################################
//------------------------------------------------------------------------------
/// Computes new values for Pos, Check, Vel and Ros (using Verlet).
/// The value of Vel always set to be reset.
///
/// Calcula nuevos valores de  Pos, Check, Vel y Rhop (usando Verlet).
/// El valor de Vel para bound siempre se pone a cero.
//------------------------------------------------------------------------------
template<bool floating,bool shift> __global__ void KerComputeStepVerlet
  (unsigned n,unsigned npb,float rhopzero,float rhopoutmin,float rhopoutmax
  ,const float4 *velrhop1,const float4 *velrhop2
  ,const float *ar,const float3 *ace,const float4 *shiftposfs
  ,double dt,double dt205,double dt2,float3 gravity
  ,double2 *movxy,double *movz,typecode *code,float4 *velrhopnew)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    if(p<npb){ //-Particles: Fixed & Moving.
      float rrhop=float(double(velrhop2[p].w)+dt2*ar[p]);
      rrhop=(rrhop<rhopzero? rhopzero: rrhop); //-To prevent absorption of fluid particles by boundaries. | Evita q las boundary absorvan a las fluidas.
      velrhopnew[p]=make_float4(0,0,0,rrhop);
    }
    else{ //-Particles: Floating & Fluid.
      //-Updates density.
      float4 rvelrhop2=velrhop2[p];
      rvelrhop2.w=float(double(rvelrhop2.w)+dt2*ar[p]);
      float4 rvel1=velrhop1[p];
      if(!floating || CODE_IsFluid(code[p])){ //-Particles: Fluid.
        //-Checks rhop limits.
        if(rvelrhop2.w<rhopoutmin||rvelrhop2.w>rhopoutmax){ //-Only brands as excluded normal particles (not periodic). | Solo marca como excluidas las normales (no periodicas).
          const typecode rcode=code[p];
          if(CODE_IsNormal(rcode))code[p]=CODE_SetOutRhop(rcode);
        }
        //-Computes and stores position displacement.
        const float3 race=ace[p];
        const double acegrx=double(race.x)+gravity.x;
        const double acegry=double(race.y)+gravity.y;
        const double acegrz=double(race.z)+gravity.z;
        double dx=double(rvel1.x)*dt + acegrx*dt205;
        double dy=double(rvel1.y)*dt + acegry*dt205;
        double dz=double(rvel1.z)*dt + acegrz*dt205;
        if(shift){
          const float4 rshiftpos=shiftposfs[p];
          dx+=double(rshiftpos.x);
          dy+=double(rshiftpos.y);
          dz+=double(rshiftpos.z);
        }
        movxy[p]=make_double2(dx,dy);
        movz[p]=dz;
        //-Updates velocity.
        rvelrhop2.x=float(double(rvelrhop2.x) + acegrx*dt2);
        rvelrhop2.y=float(double(rvelrhop2.y) + acegry*dt2);
        rvelrhop2.z=float(double(rvelrhop2.z) + acegrz*dt2);
        velrhopnew[p]=rvelrhop2;
      }
      else{ //-Particles: Floating.
        rvel1.w=(rvelrhop2.w<rhopzero? rhopzero: rvelrhop2.w); //-To prevent absorption of fluid particles by boundaries. | Evita q las floating absorvan a las fluidas.
        velrhopnew[p]=rvel1;
      }
    }
  }
}
//==============================================================================
/// Updates particles according to forces and dt using Verlet. 
/// Actualizacion de particulas segun fuerzas y dt usando Verlet.
//==============================================================================
void ComputeStepVerlet(bool floating,bool shift,unsigned np,unsigned npb
  ,const float4 *velrhop1,const float4 *velrhop2
  ,const float *ar,const float3 *ace,const float4 *shiftposfs
  ,double dt,double dt2,float rhopzero,float rhopoutmin,float rhopoutmax,tfloat3 gravity
  ,typecode *code,double2 *movxy,double *movz,float4 *velrhopnew,hipStream_t stm)
{
  double dt205=(0.5*dt*dt);
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    if(shift){    const bool shift=true;
      if(floating)KerComputeStepVerlet<true ,shift> <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,rhopzero,rhopoutmin,rhopoutmax,velrhop1,velrhop2,ar,ace,shiftposfs,dt,dt205,dt2,Float3(gravity),movxy,movz,code,velrhopnew);
      else        KerComputeStepVerlet<false,shift> <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,rhopzero,rhopoutmin,rhopoutmax,velrhop1,velrhop2,ar,ace,shiftposfs,dt,dt205,dt2,Float3(gravity),movxy,movz,code,velrhopnew);
    }else{        const bool shift=false;
      if(floating)KerComputeStepVerlet<true ,shift> <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,rhopzero,rhopoutmin,rhopoutmax,velrhop1,velrhop2,ar,ace,shiftposfs,dt,dt205,dt2,Float3(gravity),movxy,movz,code,velrhopnew);
      else        KerComputeStepVerlet<false,shift> <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,rhopzero,rhopoutmin,rhopoutmax,velrhop1,velrhop2,ar,ace,shiftposfs,dt,dt205,dt2,Float3(gravity),movxy,movz,code,velrhopnew);
    }
  }
}

//------------------------------------------------------------------------------
/// Computes new values for Pos, Check, Vel and Ros (used with Symplectic-Predictor).
/// Calcula los nuevos valores de Pos, Vel y Rhop (usando para Symplectic-Predictor).
//------------------------------------------------------------------------------
template<bool floating,bool shift> __global__ void KerComputeStepSymplecticPre
  (unsigned n,unsigned npb
  ,const float4 *velrhoppre,const float *ar,const float3 *ace,const float4 *shiftposfs
  ,double dtm,float rhopzero,float rhopoutmin,float rhopoutmax,float3 gravity
  ,typecode *code,double2 *movxy,double *movz,float4 *velrhop)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    if(p<npb){ //-Particles: Fixed & Moving.
      float4 rvelrhop=velrhoppre[p];
      rvelrhop.w=float(double(rvelrhop.w)+dtm*ar[p]);
      rvelrhop.w=(rvelrhop.w<rhopzero? rhopzero: rvelrhop.w); //-To prevent absorption of fluid particles by boundaries. | Evita que las boundary absorvan a las fluidas.
      velrhop[p]=rvelrhop;
    }
    else{ //-Particles: Floating & Fluid.
      //-Updates density.
      float4 rvelrhop=velrhoppre[p];
      rvelrhop.w=float(double(rvelrhop.w)+dtm*ar[p]);
      if(!floating || CODE_IsFluid(code[p])){ //-Particles: Fluid.
        //-Checks rhop limits.
        if(rvelrhop.w<rhopoutmin||rvelrhop.w>rhopoutmax){//-Only brands as excluded normal particles (not periodic). | Solo marca como excluidas las normales (no periodicas).
          const typecode rcode=code[p];
          if(CODE_IsNormal(rcode))code[p]=CODE_SetOutRhop(rcode);
        }
        //-Computes and stores position displacement.
        double dx=double(rvelrhop.x)*dtm;
        double dy=double(rvelrhop.y)*dtm;
        double dz=double(rvelrhop.z)*dtm;
        if(shift){
          const float4 rshiftpos=shiftposfs[p];
          dx+=double(rshiftpos.x);
          dy+=double(rshiftpos.y);
          dz+=double(rshiftpos.z);
        }
        movxy[p]=make_double2(dx,dy);
        movz[p]=dz;
        //-Updates velocity.
        const float3 race=ace[p];
        rvelrhop.x=float(double(rvelrhop.x) + (double(race.x)+gravity.x) * dtm);
        rvelrhop.y=float(double(rvelrhop.y) + (double(race.y)+gravity.y) * dtm);
        rvelrhop.z=float(double(rvelrhop.z) + (double(race.z)+gravity.z) * dtm);
      }
      else{ //-Particles: Floating.
        rvelrhop.w=(rvelrhop.w<rhopzero? rhopzero: rvelrhop.w); //-To prevent absorption of fluid particles by boundaries. | Evita q las floating absorvan a las fluidas.
      }
      //-Stores new velocity and density.
      velrhop[p]=rvelrhop;
    }
  }
}
//==============================================================================
/// Updates particles using Symplectic-Predictor.
/// Actualizacion de particulas usando Symplectic-Predictor.
//==============================================================================   
void ComputeStepSymplecticPre(bool floating,bool shift,unsigned np,unsigned npb
  ,const float4 *velrhoppre,const float *ar,const float3 *ace,const float4 *shiftposfs
  ,double dtm,float rhopzero,float rhopoutmin,float rhopoutmax,tfloat3 gravity
  ,typecode *code,double2 *movxy,double *movz,float4 *velrhop,hipStream_t stm)
{
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    if(shift){    const bool shift=true;
      if(floating)KerComputeStepSymplecticPre<true ,shift> <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,velrhoppre,ar,ace,shiftposfs,dtm,rhopzero,rhopoutmin,rhopoutmax,Float3(gravity),code,movxy,movz,velrhop);
      else        KerComputeStepSymplecticPre<false,shift> <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,velrhoppre,ar,ace,shiftposfs,dtm,rhopzero,rhopoutmin,rhopoutmax,Float3(gravity),code,movxy,movz,velrhop);
    }else{        const bool shift=false;
      if(floating)KerComputeStepSymplecticPre<true ,shift> <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,velrhoppre,ar,ace,shiftposfs,dtm,rhopzero,rhopoutmin,rhopoutmax,Float3(gravity),code,movxy,movz,velrhop);
      else        KerComputeStepSymplecticPre<false,shift> <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,velrhoppre,ar,ace,shiftposfs,dtm,rhopzero,rhopoutmin,rhopoutmax,Float3(gravity),code,movxy,movz,velrhop);
    }
  }
}

//------------------------------------------------------------------------------
/// Computes new values for Pos, Check, Vel and Ros (using Verlet).
/// The value of Vel always set to be reset.
///
/// Calcula los nuevos valores de Pos, Vel y Rhop (usandopara Symplectic-Corrector).
/// Pone vel de contorno a cero.
//------------------------------------------------------------------------------
template<bool floating,bool shift> __global__ void KerComputeStepSymplecticCor
  (unsigned n,unsigned npb
  ,const float4 *velrhoppre,const float *ar,const float3 *ace,const float4 *shiftposfs
  ,double dtm,double dt,float rhopzero,float rhopoutmin,float rhopoutmax,float3 gravity
  ,typecode *code,double2 *movxy,double *movz,float4 *velrhop)
{
  unsigned p=blockIdx.x*blockDim.x + threadIdx.x; //-Number of particle.
  if(p<n){
    if(p<npb){ //-Particles: Fixed & Moving.
      double epsilon_rdot=(-double(ar[p])/double(velrhop[p].w))*dt;
      float rrhop=float(double(velrhoppre[p].w) * (2.-epsilon_rdot)/(2.+epsilon_rdot));
      rrhop=(rrhop<rhopzero? rhopzero: rrhop); //-To prevent absorption of fluid particles by boundaries. | Evita q las boundary absorvan a las fluidas.
      velrhop[p]=make_float4(0,0,0,rrhop);
    }
    else{ //-Particles: Floating & Fluid.
      //-Updates density.
      double epsilon_rdot=(-double(ar[p])/double(velrhop[p].w))*dt;
      float4 rvelrhop=velrhoppre[p];
      rvelrhop.w=float(double(rvelrhop.w) * (2.-epsilon_rdot)/(2.+epsilon_rdot));
      if(!floating || CODE_IsFluid(code[p])){//-Particles: Fluid.
        float4 rvelp=rvelrhop;
        //-Updates velocity.
        float3 race=ace[p];
        rvelrhop.x=float(double(rvelrhop.x) + (double(race.x)+gravity.x) * dt);
        rvelrhop.y=float(double(rvelrhop.y) + (double(race.y)+gravity.y) * dt);
        rvelrhop.z=float(double(rvelrhop.z) + (double(race.z)+gravity.z) * dt);
        //-Checks rhop limits.
        if(rvelrhop.w<rhopoutmin||rvelrhop.w>rhopoutmax){//-Only brands as excluded normal particles (not periodic). | Solo marca como excluidas las normales (no periodicas).
          const typecode rcode=code[p];
          if(CODE_IsNormal(rcode))code[p]=CODE_SetOutRhop(rcode);
        }
        //-Computes and stores position displacement.
        double dx=(double(rvelp.x)+double(rvelrhop.x)) * dtm;
        double dy=(double(rvelp.y)+double(rvelrhop.y)) * dtm;
        double dz=(double(rvelp.z)+double(rvelrhop.z)) * dtm;
        if(shift){
          const float4 rshiftpos=shiftposfs[p];
          dx+=double(rshiftpos.x);
          dy+=double(rshiftpos.y);
          dz+=double(rshiftpos.z);
        }
        movxy[p]=make_double2(dx,dy);
        movz[p]=dz;
      }
      else{ //-Particles: Floating.
        rvelrhop.w=(rvelrhop.w<rhopzero? rhopzero: rvelrhop.w); //-To prevent absorption of fluid particles by boundaries. | Evita q las floating absorvan a las fluidas.
      }
      //-Stores new velocity and density.
      velrhop[p]=rvelrhop;
    }
  }
}
//==============================================================================
/// Updates particles using Symplectic-Corrector.
/// Actualizacion de particulas usando Symplectic-Corrector.
//==============================================================================   
void ComputeStepSymplecticCor(bool floating,bool shift,unsigned np,unsigned npb
  ,const float4 *velrhoppre,const float *ar,const float3 *ace,const float4 *shiftposfs
  ,double dtm,double dt,float rhopzero,float rhopoutmin,float rhopoutmax,tfloat3 gravity
  ,typecode *code,double2 *movxy,double *movz,float4 *velrhop,hipStream_t stm)
{
  if(np){
    dim3 sgrid=GetSimpleGridSize(np,SPHBSIZE);
    if(shift){    const bool shift=true;
      if(floating)KerComputeStepSymplecticCor<true,shift>  <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,velrhoppre,ar,ace,shiftposfs,dtm,dt,rhopzero,rhopoutmin,rhopoutmax,Float3(gravity),code,movxy,movz,velrhop);
      else        KerComputeStepSymplecticCor<false,shift> <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,velrhoppre,ar,ace,shiftposfs,dtm,dt,rhopzero,rhopoutmin,rhopoutmax,Float3(gravity),code,movxy,movz,velrhop);
    }else{        const bool shift=false;
      if(floating)KerComputeStepSymplecticCor<true,shift>  <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,velrhoppre,ar,ace,shiftposfs,dtm,dt,rhopzero,rhopoutmin,rhopoutmax,Float3(gravity),code,movxy,movz,velrhop);
      else        KerComputeStepSymplecticCor<false,shift> <<<sgrid,SPHBSIZE,0,stm>>> (np,npb,velrhoppre,ar,ace,shiftposfs,dtm,dt,rhopzero,rhopoutmin,rhopoutmax,Float3(gravity),code,movxy,movz,velrhop);
    }
  }
}


}


